#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <random>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// from https://github.com/jarro2783/cxxopts
#include "cxxopts.hpp"

#define cudaCheck(err) (cudaErrorCheck(err, __FILE__, __LINE__))
#define cublasCheck(err) (cublasErrorCheck(err, __FILE__, __LINE__))
#define ROUND_UP_TO_NEAREST(M, N) (((M) + (N)-1) / (N))

enum Algo
{
    cublas = 0,
    basic,
    gmem_coalesced,
    smem,
    smem_multioutput,
    numAlgos
};

const char *algo2str(Algo a)
{
    switch (a)
    {
    case cublas:
        return "cublas";
    case basic:
        return "basic";
    case gmem_coalesced:
        return "gmem_coalesced";
    case smem:
        return "sharedmem";
    case smem_multioutput:
        return "sharedmem_multioutput";
    default:
        return "INVALID";
    }
}

void cudaErrorCheck(hipError_t error, const char *file, int line);
void cublasErrorCheck(hipblasStatus_t status, const char *file, int line);
void randomize_matrix(float *mat, int N);
void const_init_matrix(float *mat, int N, float F);
bool verify_matrix(float *expected, float *actual, int M, int N);
void print_matrix(const float *A, int M, int N, std::ostream &outs);
void runAlgo(Algo algo, hipblasHandle_t handle, int M, int N, int K, float alpha, float *A, float *B, float beta, float *C);
void runCublas(hipblasHandle_t handle, int M, int N, int K, float alpha, float *A, float *B, float beta, float *C);

const std::string errLogFile = "gemmValidationFailure.txt";

// NB: must use a single generator to avoid duplicates
std::default_random_engine generator(2);
std::uniform_real_distribution<float> distribution(0, 1);

int main(int argc, char **argv)
{
    // command-line flags
    cxxopts::Options options("gemm.cu", "CUDA GEMM kernels");
    options.add_options()("size", "matrix size (N x N)", cxxopts::value<uint16_t>()->default_value("128"))                //
        ("reps", "repeat GEMM this many times", cxxopts::value<uint16_t>()->default_value("1"))                           //
        ("algo", "GEMM algorithm to use, a number in [0,4], 0 is cuBLAS", cxxopts::value<uint16_t>()->default_value("0")) //
        ("validate", "Validate output against cuBLAS", cxxopts::value<bool>()->default_value("true"))                     //
        ("rngseed", "PRNG seed", cxxopts::value<uint>()->default_value("2"))                     //
        ("h,help", "Print usage");

    auto clFlags = options.parse(argc, argv);
    if (clFlags.count("help"))
    {
        std::cout << options.help() << std::endl;
        exit(0);
    }
    const uint16_t SIZE = clFlags["size"].as<uint16_t>();
    if (SIZE % 32 != 0)
    {
        //std::cout << "--size must be a multiple of 32" << std::endl;
        //exit(EXIT_FAILURE);
    }
    const uint16_t REPS = clFlags["reps"].as<uint16_t>();
    const Algo ALGO = static_cast<Algo>(clFlags["algo"].as<uint16_t>());
    if (ALGO >= numAlgos)
    {
        printf("Invalid algorithm: %d\n", ALGO);
        exit(EXIT_FAILURE);
    }

    const bool VALIDATE = clFlags["validate"].as<bool>();
    const uint SEED = clFlags["rngseed"].as<uint>();
    generator.seed(SEED);
    printf("Multiplying two %u x %u matrices with %u trials using %s algorithm\n", SIZE, SIZE, REPS, algo2str(ALGO));

    cudaCheck(hipSetDevice(0));

    // Setup cublas
    hipblasHandle_t handle;
    cublasCheck(hipblasCreate(&handle));

    // Using cudaEvent for gpu stream timing, cudaEvent is equivalent to
    // publishing event tasks in the target stream
    hipEvent_t beg, end;
    cudaCheck(hipEventCreate(&beg));
    cudaCheck(hipEventCreate(&end));

    uint16_t m = SIZE, n = SIZE, k = SIZE;

    // GEMM computes C = α*AB+β*C

    // just do pure A*B (for simpler debugging)
    float alpha = 1.0, beta = 1.0, initC = 1.0;

    float *A = nullptr, *B = nullptr, *C = nullptr, *C_ref = nullptr;     // host matrices
    float *dA = nullptr, *dB = nullptr, *dC = nullptr, *dC_ref = nullptr; // device matrices

    A = (float *)malloc(sizeof(float) * SIZE * SIZE);
    B = (float *)malloc(sizeof(float) * SIZE * SIZE);
    C = (float *)malloc(sizeof(float) * SIZE * SIZE);
    C_ref = (float *)malloc(sizeof(float) * SIZE * SIZE);

    randomize_matrix(A, SIZE * SIZE);
    randomize_matrix(B, SIZE * SIZE);
    randomize_matrix(C, SIZE * SIZE);

    const_init_matrix(C, SIZE * SIZE, initC);
    // print_matrix(A, SIZE, SIZE, std::cout);
    // print_matrix(B, SIZE, SIZE, std::cout);
    // print_matrix(C, SIZE, SIZE, std::cout);

    cudaCheck(hipMalloc((void **)&dA, sizeof(float) * SIZE * SIZE));
    cudaCheck(hipMalloc((void **)&dB, sizeof(float) * SIZE * SIZE));
    cudaCheck(hipMalloc((void **)&dC, sizeof(float) * SIZE * SIZE));
    cudaCheck(hipMalloc((void **)&dC_ref, sizeof(float) * SIZE * SIZE));

    cudaCheck(hipMemcpy(dA, A, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dB, B, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC, C, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC_ref, C, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice));

    printf("dimensions(m=n=k) %u, alpha: %f, beta: %f\n", m, alpha, beta);

    // Verify the correctness of the calculation, and execute it once before the
    // kernel function timing to avoid cold start errors
    if (!VALIDATE)
    {
        printf("disabled validation\n");
    }
    else
    {
        // run cublas to get correct answer in dC_ref
        runCublas(handle, m, n, k, alpha, dA, dB, beta, dC_ref);

        // run user's algorithm, filling in dC
        runAlgo(ALGO, handle, m, n, k, alpha, dA, dB, beta, dC);

        cudaCheck(hipDeviceSynchronize());

        // copy both results back to host
        hipMemcpy(C, dC, sizeof(float) * m * n, hipMemcpyDeviceToHost);
        hipMemcpy(C_ref, dC_ref, sizeof(float) * m * n, hipMemcpyDeviceToHost);

        if (verify_matrix(C_ref, C, n, m))
        {
            printf("Validated successfully!\n");
        }
        else
        {
            printf("Failed validation against NVIDIA cuBLAS.\n");
            std::cout << " Logging faulty output into " << errLogFile << "\n";
            std::ofstream fs;
            fs.open(errLogFile, std::ios::out | std::ios::trunc);
            fs << "α=" << alpha << " β=" << beta << std::endl;
            fs << "C matrix initialized to " << initC << std::endl << std::endl;
            fs << "A:" << std::endl;
            print_matrix(A, m, n, fs);
            fs << "B:" << std::endl;
            print_matrix(B, m, n, fs);
            fs << "C:" << std::endl;
            print_matrix(C, m, n, fs);
            fs << "Expected:" << std::endl;
            print_matrix(C_ref, m, n, fs);
            fs.close();
            exit(EXIT_FAILURE);
        }
    }

    // timing run(s)
    hipEventRecord(beg);
    for (int j = 0; j < REPS; j++)
    {
        // We don't reset dC between runs to save time
        runAlgo(ALGO, handle, m, n, k, alpha, dA, dB, beta, dC);
        cudaCheck(hipDeviceSynchronize());
    }

    // TODO: measure timing without memory transfers?
    cudaCheck(hipEventRecord(end));
    cudaCheck(hipEventSynchronize(beg));
    cudaCheck(hipEventSynchronize(end));
    float elapsed_time;
    cudaCheck(hipEventElapsedTime(&elapsed_time, beg, end));
    elapsed_time /= 1000.; // Convert to seconds

    double flops = (double)2 * m * n * k;
    printf(
        "Average elapsed time: (%7.6f) s, performance: (%7.2f) GFLOPS. size: (%u).\n",
        elapsed_time / REPS,
        (REPS * flops * 1e-9) / elapsed_time,
        m);

    // free CPU and GPU memory
    free(A);
    free(B);
    free(C);
    free(C_ref);
    cudaCheck(hipFree(dA));
    cudaCheck(hipFree(dB));
    cudaCheck(hipFree(dC));
    cudaCheck(hipFree(dC_ref));
    cublasCheck(hipblasDestroy(handle));

    return 0;
}

/** Function to check for errors in CUDA API calls */
void cudaErrorCheck(hipError_t error, const char *file, int line)
{
    if (error != hipSuccess)
    {
        printf("[CUDA ERROR] at file %s:%d:\n%s: %s\n", file, line,
               hipGetErrorName(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
};

void cublasErrorCheck(hipblasStatus_t status, const char *file, int line)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("[CUDA ERROR] at file %s:%d:\n %s: %s\n", file, line,
               cublasGetStatusName(status), cublasGetStatusString(status));
        exit(EXIT_FAILURE);
    }
}

/** Initialize the given matrix `mat` which has `N` contiguous values. Contents of `mat` are set to random values. */
void randomize_matrix(float *mat, int N)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = distribution(generator);
    }
}

void const_init_matrix(float *mat, int N, float F)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = F;
    }
}

/** Print the given MxN matrix `mat` to the provided output stream. */
void print_matrix(const float *A, int M, int N, std::ostream &outs)
{
    outs << "[";
    for (int i = 0; i < M * N; i++)
    {
        if ((i + 1) % N == 0)
        {
            outs << std::fixed << std::setprecision(3) << A[i];
        }
        else
        {
            outs << std::fixed << std::setprecision(3) << A[i] << ", ";
        }
        if ((i + 1) % N == 0)
        {
            if (i + 1 < M * N)
                outs << ";" << std::endl;
        }
    }
    outs << "]" << std::endl << std::endl;
}

bool verify_matrix(float *expected, float *actual, int M, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < M; j++)
        {
            float fexp = (expected[(i * N) + j]);
            float fact = (actual[(i * N) + j]);
            double diff = std::fabs(fexp - fact);
            if (diff > 0.002)
            {
                printf("Divergence! Should be %5.3f, is %5.3f (diff %5.3f) at [%d,%d]\n",
                       fexp, fact, diff, i, j);
                return false;
            }
        }
    }
    return true;
}

void runCublas(hipblasHandle_t handle, int M, int N, int K, float alpha,
               float *A, float *B, float beta, float *C)
{
    // cuBLAS uses *column-major* order. So we change the order of our row-major A &
    // B, since (B^T*A^T)^T = (A*B)
    // hipblasStatus_t ok = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_16F,
    //                                  N, A, HIP_R_16F, K, &beta, C, HIP_R_16F, N, /*HIPBLAS_COMPUTE_16F*/ HIPBLAS_COMPUTE_16F_PEDANTIC,
    //                                  HIPBLAS_GEMM_DEFAULT);
    hipblasStatus_t ok = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, N, A, K, &beta, C, N);
    cublasCheck(ok);
}

__global__ void runBasic(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
    const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < M && y < N)
    {
        float tmp = 0.0;
        // C = α*(AxB)+β*C
        for (int i = 0; i < K; ++i)
        {
            // tmp += __A__[x][i] * __B__[i][y]
            tmp += A[(x * K) + i] * B[(i * N) + y];
        }
        // __C__[x][y]
        C[(x * N) + y] = (alpha * tmp) + (beta * C[x * N + y]);
    }
}

__global__ void runGmemCoalesced(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
    const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < N && y < M)
    {
        float tmp = 0.0;
        // C = α*(AxB)+β*C
        for (int i = 0; i < K; ++i)
        {
            tmp += A[(y * K) + i] * B[(i * N) + x];
        }
        C[(y * N) + x] = (alpha * tmp) + (beta * C[y * N + x]);
    }
}

const uint F = 32;

__global__ void runSharedMem(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
    __shared__ float SA[F][F];
    __shared__ float SB[F][F];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0;

    for (int i = 0; i < N; i += F) {
        SA[threadIdx.y][threadIdx.x] = A[row * N + i + threadIdx.x];
        SB[threadIdx.y][threadIdx.x] = B[(i + threadIdx.y) * N + col];
        __syncthreads();

        for (int t = 0; t < F; ++t) {
            sum += SA[threadIdx.y][t] * SB[t][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < N && col < M) {
        // We already assert that SIZE % F == 0, 
        // so there will not be any corner cases.
        // int upper = (N / F) * F;
        // for (int i = 0; i < N % F; ++i) {
        //     sum += A[row * N + upper + i] * B[(upper + i) * N + col];
        // }
        C[row * N + col] = alpha * sum + beta * C[row * N + col];
    }
}

const uint G = 4;

__global__ void runSharedMemMultiOutput(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
    // HW3 TODO: Copy your runSharedMem() code here and update it so that each thread computes the result for GxG cells 
    // of the output matrix C. Each thread should accumulate temporary results in the local LC matrix, provided below,
    // before writing them to C in global memory.

    // Note, you will also need to change the grid dimensions in the kernel launch below. You should experiment 
    // with different values of F and G to see how they affect performance.

    __shared__ float SA[F][F];
    __shared__ float SB[F][F];

    float LC[G][G] = {0.0};

}

void runAlgo(Algo algo, hipblasHandle_t handle, int M, int N, int K, float alpha,
             float *A, float *B, float beta, float *C)
{
    switch (algo)
    {
    case cublas:
        runCublas(handle, M, N, K, alpha, A, B, beta, C);
        break;
    case basic:
    {
        dim3 gridDim(ROUND_UP_TO_NEAREST(M, 32), ROUND_UP_TO_NEAREST(N, 32));
        dim3 blockDim(32, 32);
        runBasic<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
        break;
    }
    case gmem_coalesced:
    {
        dim3 gridDim(ROUND_UP_TO_NEAREST(M, 32), ROUND_UP_TO_NEAREST(N, 32));
        dim3 blockDim(32, 32);
        runGmemCoalesced<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
        break;
    }
    case smem:
    {
        assert(0 == M % F);
        assert(0 == N % F);
        assert(0 == K % F);
        dim3 gridDim(ROUND_UP_TO_NEAREST(M, F), ROUND_UP_TO_NEAREST(N, F));
        dim3 blockDim(F, F);
        runSharedMem<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
        break;
    }
    case smem_multioutput:
    {
        assert(0 == M % F);
        assert(0 == N % F);
        assert(0 == K % F);
        assert(0 == F % G);
        assert((F*F) / (G*G) >= F);
        // TODO: update your grid here
        dim3 gridDim(ROUND_UP_TO_NEAREST(M, 32), ROUND_UP_TO_NEAREST(N, 32));
        dim3 blockDim(32, 32);
        runSharedMemMultiOutput<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
        break;
    }
    default:
        printf("Invalid algorithm: %d\n", algo);
        exit(EXIT_FAILURE);
    }
    cudaCheck(hipDeviceSynchronize()); // wait for kernel to finish
    cudaCheck(hipGetLastError());      // check for errors from kernel run
}
